// To measure effects of memory coalescing. Coalescing.cu 
// B. Wilkinson Jan 30, 2011

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define BlockSize 16 				// Size of blocks, 32 x 32 threads, fixed, used globally

__global__ void gpu_Comput (int *h, int N, int T) {

// Array loaded with global thread ID that acesses that location

	int col = threadIdx.x + blockDim.x * blockIdx.x; 
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	int threadID = col + row * N;
	int index = row + col * N;		// sequentially down each row

	for (int t = 0; t < T; t++)		// loop to repeat to reduce other time effects
	   h[index] = threadID;  		// load array with flattened global thread ID
}

void printArray(int *h, int N) {

	printf("Results of computation, every N/8 numbers, eight numbers\n");

	for (int row = 0; row < N; row += N/8) {
	  for (int col = 0; col < N; col += N/8) 
	 	printf("%6d  ", h[col + row * N]);
	  printf("\n"); 
	}
}

int main(int argc, char *argv[])  {

	int T = 1000;				 	// number of iterations, entered at keyboard
	int B = 128;				 	// number of blocks, entered at keyboard
	char key;

	int *h, *dev_h;				// ptr to array holding numbers on host and device

	hipEvent_t start, stop;     		// cuda events to measure time
	float elapsed_time_ms1;
	hipEventCreate( &start ); 
	hipEventCreate( &stop );

/* ------------------------- Keyboard input -----------------------------------*/

for(int i=0;i<50;i++) {  // loop to repeat complete program	

	//printf("Grid Structure 2-D grid, 2-D blocks\n");
//	printf("Blocks fixed at 16 x 16 threads, 512 threads, max for compute cap. 1.x\n");
//	printf("Enter number of blocks in grid, each dimension, currently %d\n",B);
//	scanf("%d",&B);
//	printf("Enter number of iterations, currently %d\n",T);
//	scanf("%d",&T);

	int N = B * BlockSize;				// size of data array, given input data

	printf("Array size (and total grid-block size) %d x %d\n", N, N);

	dim3 Block(BlockSize, BlockSize);			//Block structure, 32 x 32 max
	dim3 Grid(B, B);					//Grid structure, B x B	

/* ------------------------- Allocate Memory-----------------------------------*/

	int size = N * N * sizeof(int);		// number of bytes in total in array
	h = (int*) malloc(size);			// Array on host

	hipMalloc((void**)&dev_h, size);		// allocate device memory

/* ------------------------- GPU Computation -----------------------------------*/
			
	hipEventRecord( start, 0 );
	
	gpu_Comput<<< Grid, Block >>>(dev_h, N, T);
	
         hipMemcpy(h,dev_h, size ,hipMemcpyDeviceToHost);  //Get results to check

	hipEventRecord( stop, 0 );     		// instrument code to measue end time
	hipEventSynchronize( stop );		// wait for all work done by threads
	hipEventElapsedTime( &elapsed_time_ms1, start, stop );

//	cudaMemcpy(h,dev_h, size ,cudaMemcpyDeviceToHost);  //Get results to check

	printArray(h,N);
	printf("\nTime to calculate results on GPU: %f ms.\n", elapsed_time_ms1);

/* -------------------------REPEAT PROGRAM INPUT-----------------------------------*/

	//printf("\nEnter c to repeat, return to terminate\n");

//	scanf("%c",&key);
//	scanf("%c",&key);

} 
 
/* --------------  clean up  ---------------------------------------*/
	
	free(h);
	hipFree(dev_h);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
  
	return 0;
}


