
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void EmptyKernel() { }

int main() {

    const int N = 100000;

    float time, cumulative_time = 0.f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int i=0; i<N; i++) { 

        hipEventRecord(start, 0);
        EmptyKernel<<<1,1>>>(); 
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        cumulative_time = cumulative_time + time;

    }

    printf("Kernel launch overhead time:  %3.5f ms \n", cumulative_time / N);
    return 0;
}
